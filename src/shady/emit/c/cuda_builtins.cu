#include "hip/hip_runtime.h"
__shared__ uvec3 __shady_make_thread_local(RealGlobalInvocationId);
__shared__ uvec3 __shady_make_thread_local(RealLocalInvocationId);

#define GlobalInvocationId __shady_thread_local_access(RealGlobalInvocationId)
#define LocalInvocationId __shady_thread_local_access(RealLocalInvocationId)

__device__ void __shady_prepare_builtins() {
    LocalInvocationId.arr[0] = threadIdx.x;
    LocalInvocationId.arr[1] = threadIdx.y;
    LocalInvocationId.arr[2] = threadIdx.z;
    GlobalInvocationId.arr[0] = threadIdx.x + blockDim.x * blockIdx.x;
    GlobalInvocationId.arr[1] = threadIdx.y + blockDim.y * blockIdx.y;
    GlobalInvocationId.arr[2] = threadIdx.z + blockDim.z * blockIdx.z;
}
